/*
 * @author Marc Suchard
 */
#ifndef _Included_SharedFunctionsKernel
#define _Included_SharedFunctionsKernel

/**************INCLUDES***********/
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include "CUDASharedFunctions.h"

/**************CODE***********/
#ifdef __cplusplus
extern "C" {
#endif

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                             hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

void getGPUInfo(int iDevice, char *name, int *memory, int *speed) {
	hipDeviceProp_t deviceProp;
	memset(&deviceProp, 0, sizeof(deviceProp));
	hipGetDeviceProperties(&deviceProp, iDevice);
	*memory = deviceProp.totalGlobalMem;
	*speed = deviceProp.clockRate;
	strcpy(name, deviceProp.name);
}

#ifdef __cplusplus
}
#endif
#endif

